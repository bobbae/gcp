
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void say_hello() {
	printf("Hello world from the GPU!\n");
}

int main() {
	printf("Hello world from the CPU!\n");

	say_hello<<<1,1>>>();
	hipDeviceSynchronize();

	return 0;
}
